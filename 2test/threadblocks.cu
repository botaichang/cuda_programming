
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ 
void hello()
{
   printf("hello world, I am a thread in block %d\n",blockIdx.x);
}


int main(int argc, char **argv)
{
   // lauch the kernel
   hello<<<NUM_BLOCKS,BLOCK_WIDTH>>>();
    
   //force the printf() to flush
   hipDeviceSynchronize();

   printf("That's all!\n");
   return 0;
}
