#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */

__global__ void helloFromGPU(int count)
{
    for (int i=blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x  )
    {
    printf("gridDim.x,%d | blockDim.x %d | blockIdx.x %d | threadIdx.x %d\n",gridDim.x,blockDim.x,blockIdx.x,threadIdx.x);
    printf("%d, Hello World from GPU!,blockIdx: %d,threadIdx:%d \n", i,blockIdx.x, threadIdx.x);
    }
}



int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<5, 10>>>(50);
    
    CHECK(hipDeviceReset());
    return 0;
}


