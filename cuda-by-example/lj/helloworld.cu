
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("hello world from GPU (block thread)%d,%d!\n",blockIdx.x,threadIdx.x);
	printf("blockdim %d,%d,%d\n",blockDim.x,blockDim.y,blockDim.z);
	printf("griddim %d,%d,%d\n",gridDim.x,gridDim.y,gridDim.z);
}

int main(int argc, char const *argv[])
{

	printf("hello world from cpu\n");

    dim3 grid(3);
    dim3 block(2);
	printf("block %d,%d,%d\n",block.x,block.y,block.z);
	printf("grid %d,%d,%d\n",grid.x,grid.y,grid.z);

	helloFromGPU<<<grid,block>>>();
	hipDeviceReset();
	//cudaDeviceSynchronize();
	return 0;
}
